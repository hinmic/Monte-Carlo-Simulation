#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "helper_image.h"
#include "helper_string.h"
#include "helper_timer.h"
#include "exception.h"

int BLOCKSIZE = 16;	        // number of threads in each block

int N = 2;

int NUMTRIALS = N * 1024;   // size of the array

// ranges for the random numbers:
const float GMIN = 20.0;    // ground distance in meters
const float GMAX = 30.0;    // ground distance in meters
const float HMIN = 10.0;    // cliff height in meters
const float HMAX = 40.0;    // cliff height in meters
const float DMIN = 10.0;    // distance to castle in meters
const float DMAX = 20.0;    // distance to castle in meters
const float VMIN = 30.0;    // initial cannonball velocity in meters / sec
const float VMAX = 50.0;    // initial cannonball velocity in meters / sec
const float THMIN = 70.0;   // cannonball launch angle in degrees
const float THMAX = 80.0;   // cannonball launch angle in degrees

// degrees-to-radians -- callable from the device:
__device__	float Radians(float d)
{
	return (M_PI / 180.f) * d;
}

// function prototypes:
void CudaCheckError();
void TimeOfDaySeed();
float Ranf(float, float);
int Ranf(int, int);

// the kernel:
__global__	void MonteCarlo(float* dvs, float* dths, float* dgs, float* dhs, float* dds, int* dhits)
{
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    const float GRAVITY = -9.8; // acceleration due to gravity in meters / sec^2
    const float TOL = 5.0;      // tolerance in cannonball hitting the castle in meters

	// randomize everything:
	float v		= dvs[gid];
	float thr	= Radians(dths[gid]);
	float vx	= v * cos(thr);
	float vy	= v * sin(thr);
	float g		= dgs[gid];
	float h		= dhs[gid];
	float d		= dds[gid];

	int numHits = 0;

	// see if the ball doesn't even reach the cliff:
	float t = -vy / (0.5 * GRAVITY);
	float x = vx * t;
	if (x > g)
	{
        // see if the ball hits the vertical cliff face:
        t = g / vx;
        float y = vy * t + 0.5 * GRAVITY * (t * t);
        if (y > h)
        {
            // the ball hits the upper deck:
            // the time solution for this is a quadratic equation of the form:
            // at^2 + bt + c = 0.
            // where 'a' multiplies time^2
            //       'b' multiplies time
            //       'c' is a constant
            float a = 0.5 * GRAVITY;
            float b = vy;
            float c = -h;
            float disc = b * b - 4.f * a * c;   // quadratic formula discriminant

            // successfully hits the ground above the cliff:
            // get the intersection:
            disc = sqrtf(disc);
            float t1 = (-b + disc) / (2.f * a);	// time to intersect high ground
            float t2 = (-b - disc) / (2.f * a);	// time to intersect high ground

            // only care about the second intersection
            float tmax = t1;
            if (t2 > t1)
                tmax = t2;

            // how far does the ball land horizontlly from the edge of the cliff?
            float upperDist = vx * tmax - g;

            // see if the ball hits the castle:
            if (fabs(upperDist - d) <= TOL)
            {
                numHits = 1;
            }
        }   // if ball clears the cliff face
	}   // if ball gets as far as the cliff face

	dhits[gid] = numHits;
}

// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT


// main program:

int
main(int argc, char* argv[])
{
    if (argc >= 2)
        BLOCKSIZE = atoi(argv[1]);
    if (argc >= 3)
        NUMTRIALS = atoi(argv[2]) * 1024;

    int NUMBLOCKS = NUMTRIALS / BLOCKSIZE;

    TimeOfDaySeed();

    int dev = findCudaDevice(argc, (const char**)argv);

    // better to define these here so that the rand() calls don't get into the thread timing:
    float* hvs = new float[NUMTRIALS];
    float* hths = new float[NUMTRIALS];
    float* hgs = new float[NUMTRIALS];
    float* hhs = new float[NUMTRIALS];
    float* hds = new float[NUMTRIALS];
    int* hhits = new int[NUMTRIALS];

    // fill the random-value arrays:
    for (int i = 0; i < NUMTRIALS; i++)
    {
        hvs[i] = Ranf(VMIN, VMAX);
        hths[i] = Ranf(THMIN, THMAX);
        hgs[i] = Ranf(GMIN, GMAX);
        hhs[i] = Ranf(HMIN, HMAX);
        hds[i] = Ranf(DMIN, DMAX);
    }

    // allocate device memory:
    float   *dvs, *dths, *dgs, *dhs, *dds;
    int     *dhits;

    hipMalloc(&dvs, NUMTRIALS * sizeof(float));
    hipMalloc(&dths, NUMTRIALS * sizeof(float));
    hipMalloc(&dgs, NUMTRIALS * sizeof(float));
    hipMalloc(&dhs, NUMTRIALS * sizeof(float));
    hipMalloc(&dds, NUMTRIALS * sizeof(float));
    hipMalloc(&dhits, NUMTRIALS * sizeof(int));
    CudaCheckError();

    // copy host memory to the device:
    hipMemcpy(dvs, hvs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dths, hths, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dgs, hgs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dhs, hhs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dds, hds, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
    CudaCheckError();

    // setup the execution parameters:
    dim3 grid(NUMBLOCKS, 1, 1);
    dim3 threads(BLOCKSIZE, 1, 1);

    // allocate cuda events that we'll use for timing:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    CudaCheckError();

    // let the gpu go quiet:
    hipDeviceSynchronize();

    // record the start event:
    hipEventRecord(start, NULL);
    CudaCheckError();

    // execute the kernel:
    MonteCarlo <<< grid, threads >>> (IN dvs, IN dths, IN dgs, IN dhs, IN dds, OUT dhits);

    // record the stop event:
    hipEventRecord(stop, NULL);
    CudaCheckError();

    // wait for the stop event to complete:
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    CudaCheckError();

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    CudaCheckError();

    // compute the performance
    double secTotal = msecTotal / 1000;
    double megaTrialsPerSecond = (double)NUMTRIALS / secTotal / 1000000.;

    // copy result from the device to the host:
    hipMemcpy(hhits, dhits, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost);
    CudaCheckError();

    // add up the hhits[ ] array: :
    int numHits = 0;
    for (int i = 0; i < NUMTRIALS; i++)
    {
        numHits += hhits[i];
    }

    // compute the probability:
    float probability = (float)numHits / (float)(NUMTRIALS);

    // print the result:
    fprintf(stderr, "%2d threads/block : %8d trials ; probability = %6.2f%% ; megatrials/sec = %6.2lf\n", 
            BLOCKSIZE, NUMTRIALS, 100. * probability, megaTrialsPerSecond);

    // clean up host memory:
    delete [] hvs;
    delete [] hths;
    delete [] hgs;
    delete [] hhs;
    delete [] hds;
    delete [] hhits;

    // clean up device memory:
    hipFree(dvs);
    hipFree(dths);
    hipFree(dgs);
    hipFree(dhs);
    hipFree(dds);
    hipFree(dhits);
    CudaCheckError();

    return 0;
}

void
CudaCheckError()
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        fprintf(stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
    }
}

// call this if you want to force your program to use
// a different random number sequence every time you run it:
void
TimeOfDaySeed()
{
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
    y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

    time_t  timer;
    time(&timer);
    double seconds = difftime(timer, mktime(&y2k));
    unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
    srand(seed);
}

float
Ranf(float low, float high)
{
    float r = (float)rand();                // 0 - RAND_MAX
    float t = r / (float)RAND_MAX;          // 0. - 1.

    return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
    float low = (float)ilow;
    float high = ceil((float)ihigh);

    return (int)Ranf(low, high);
}